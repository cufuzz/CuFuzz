#include "hip/hip_runtime.h"
#include <fcntl.h>
#include <cmath>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <time.h>               // **[新增] 用于 wall-clock 计时**
#include "/home/fanximing/cuda-graph-llm/c_factors/mutate.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// CUDA API sequence to be tested
void UT(hipStream_t *stream, int *data, int numElements) {
    hipStreamCreateWithFlags(stream, hipStreamNonBlocking);
    hipStreamBeginCapture(*stream, hipStreamCaptureModeGlobal);

    hipStreamCaptureStatus captureStatus;
    unsigned long long id;
    hipStreamGetCaptureInfo(*stream, &captureStatus, &id);

    hipStreamEndCapture(*stream, nullptr);
}

int main() {
    // ================== 开始计时 ==================
    struct timespec start, end;                     // **[新增]**
    clock_gettime(CLOCK_MONOTONIC, &start);         // **[新增]**

    int loops = 0;
    while (true) {
        printf("%d\n", loops);                      // **[新增]**

        // initialization
        int numElements = 256;
        size_t size = numElements * sizeof(int);
        int *h_data = (int *)malloc(size);
        for (int i = 0; i < numElements; i++) {
            h_data[i] = i;
        }

        // wrap buffer
        u32 numElements_len = sizeof(numElements);
        u8 *numElements_buf = (u8 *)malloc(numElements_len);
        memcpy(numElements_buf, &numElements, numElements_len);

        // havoc
        u32 numElements_len_havoc;
        numElements_len_havoc = random_havoc(numElements_buf, numElements_len, 0);
        numElements = *(int *)numElements_buf;
        numElements %= 4096;
        if (numElements < 0) numElements = -numElements;
        if (numElements == 0) numElements += 1;

        
        FILE *file_1 = fopen("1.bin", "wb");
        fwrite(numElements_buf, numElements_len, 1, file_1);
        fclose(file_1);
        size = numElements * sizeof(int);

        // u32 h_data_len = size;
        u32 h_data_len = 4096* sizeof(int);
        u8 *h_data_buf = (u8 *)malloc(h_data_len);
        u32 h_data_len_havoc;
        h_data_len_havoc = random_havoc(h_data_buf, h_data_len, 1);
        int *temp_2 = (int *)realloc(h_data, size);
        h_data = temp_2;
        memcpy((u8 *)h_data, h_data_buf, size);
        FILE *file_2 = fopen("2.bin", "wb");
        fwrite(h_data_buf, h_data_len, 1, file_2);
        fclose(file_2);


        // ---------------- 触发条件 ----------------
        // if ( h_data[0] >0 && h_data[0] < 10) {   // **[新增]**
        //     clock_gettime(CLOCK_MONOTONIC, &end);           // **[新增]**
        //     double elapsed = (end.tv_sec  - start.tv_sec) +
        //                      (end.tv_nsec - start.tv_nsec) / 1e9;
        //     printf("Trigger condition met! numElements = %d\n", h_data[0]);
        //     printf("Wall-clock time since start: %.6f seconds\n", elapsed);
        //     exit(EXIT_SUCCESS);                             // **[新增]**
        // }
        if ( h_data[0]> 900 && h_data[0] < 1000) {   // **[新增]**
            clock_gettime(CLOCK_MONOTONIC, &end);           // **[新增]**
            double elapsed = (end.tv_sec  - start.tv_sec) +
                             (end.tv_nsec - start.tv_nsec) / 1e9;
            printf("Trigger condition met! numElements = %d\n", h_data[0]);
            printf("Wall-clock time since start: %.6f seconds\n", elapsed);
            exit(EXIT_SUCCESS);                             // **[新增]**
        }
        // -----------------------------------------


        int *d_data = nullptr;
        hipMalloc((void **)&d_data, size);

        hipStream_t stream;
        UT(&stream, h_data, numElements);

        hipStreamDestroy(stream);
        free(h_data);
        hipFree(d_data);
        free(numElements_buf);
        free(h_data_buf);
        loops++;
    }
    return EXIT_SUCCESS;
}
