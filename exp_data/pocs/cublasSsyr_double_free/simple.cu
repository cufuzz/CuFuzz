#include <fcntl.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main() {
    int loops = 0;
    srand((unsigned)time(NULL));
    while (loops < 2000) {
        
    //initialization
    printf("%d\n",loops);
    int rows = rand() % 257;
    int cols = rand() % 257;
    size_t size = rows * cols * sizeof(float);
    float *h_A = (float *)malloc(size);
    for (int i = 0; i < rows * cols; i++) {
        h_A[i] = (float)(i + 1);
    }
    //initialization

    printf("rows %d\n",rows);
    printf("cols %d\n",cols);
    printf("size %zu\n",size);


    // wrap all done!

    // Device memory pointer
    float *d_A = nullptr;

    // cuBLAS handle initialization
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Execute the CUDA API sequence

    hipMalloc((void**)&d_A, size);

        // 执行 cuBLAS 操作
    float alpha = 1.0f;
    hipblasSsyr(handle, HIPBLAS_FILL_MODE_UPPER, rows, &alpha, d_A, 1, d_A, rows);
    hipDeviceSynchronize();

    // UT(h_A, d_A, rows, cols, handle);

    // Cleanup
    free(h_A);
    hipFree(d_A);
    hipblasDestroy(handle);

   
    loops++;}
    return EXIT_SUCCESS;
}