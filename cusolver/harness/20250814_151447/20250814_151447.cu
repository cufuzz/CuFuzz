#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

// CUDA API sequence to be tested
void UT(hipsolverHandle_t *handle, hipStream_t *stream, int *d_data) {
    // 1. Free device memory
    hipFree(d_data);

    // 2. Destroy cuSolver handle
    hipsolverDnDestroy(*handle);

    // 3. Destroy CUDA stream
    hipStreamDestroy(*stream);

    // 4. Reset device
    hipDeviceReset();
}

int main() {
    // Variable definitions and initialization
    int numElements = 256;
    size_t size = numElements * sizeof(int);

    // Device memory allocation
    int *d_data = nullptr;
    hipMalloc((void**)&d_data, size);

    // cuSolver handle creation
    hipsolverHandle_t handle;
    hipsolverDnCreate(&handle);

    // Stream creation
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Execute the CUDA API sequence
    UT(&handle, &stream, d_data);

    return EXIT_SUCCESS;
}